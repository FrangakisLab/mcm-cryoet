#include "hip/hip_runtime.h"
#include "mcm.cuh"

extern "C" {
    __global__
    void mcm_kernel(const float *fg, float *ug, float ht, float3 h, dim3 dim){
        // Shared memory for this block
        __shared__ float f[10][10][10];

        // Global idx
        const unsigned int gx = blockIdx.x * blockDim.x + threadIdx.x;
        const unsigned int gy = blockIdx.y * blockDim.y + threadIdx.y;
        const unsigned int gz = blockIdx.z * blockDim.z + threadIdx.z;

        if (gx >= dim.x) return;
        if (gy >= dim.y) return;
        if (gz >= dim.z) return;

        // Local idx
        const unsigned int i = threadIdx.x+1;
        const unsigned int j = threadIdx.y+1;
        const unsigned int k = threadIdx.z+1;

        // Copy the data (direct match)
        f[i][j][k] = access_3d(fg, gx, gy, gz, dim.x, dim.y);

        // Overlap x
        if (threadIdx.x < 1){
            //printf("hit %i\n", per_idx(0-1, dim.x));
            f[i-1][j][k] = access_3d(fg, per_idx(gx-1, dim.x), gy, gz, dim.x, dim.y);
            f[i+blockDim.x][j][k] = access_3d(fg, per_idx(gx+blockDim.x, dim.x), gy, gz, dim.x, dim.y);
        }

        // Overlap y
        if (threadIdx.y < 1){
            f[i][j-1][k] = access_3d(fg, gx, per_idx(gy-1, dim.y), gz, dim.x, dim.y);
            f[i][j+blockDim.y][k] = access_3d(fg, gx, per_idx(gy+blockDim.y, dim.y), gz, dim.x, dim.y);
        }

        // Overlap z
        if (threadIdx.z < 1){
            f[i][j][k-1] = access_3d(fg, gx, gy, per_idx(gz-1, dim.z), dim.x, dim.y);
            f[i][j][k+blockDim.z] = access_3d(fg, gx, gy, per_idx(gz+blockDim.z, dim.z), dim.x, dim.y);
        }

        // Corners xy
        if (threadIdx.x < 1 && threadIdx.y < 1){
            f[i-1][j-1][k] = access_3d(fg, per_idx(gx-1, dim.x), per_idx(gy-1, dim.y), gz, dim.x, dim.y);
            f[i+blockDim.x][j+blockDim.y][k] = access_3d(fg, per_idx(gx+blockDim.x, dim.x), per_idx(gy+blockDim.y, dim.y), gz, dim.x, dim.y);
            f[i-1][j+blockDim.y][k] = access_3d(fg, per_idx(gx-1, dim.x), per_idx(gy+blockDim.y, dim.y), gz, dim.x, dim.y);
            f[i+blockDim.x][j-1][k] = access_3d(fg, per_idx(gx+blockDim.x, dim.x), per_idx(gy-1, dim.y), gz, dim.x, dim.y);
        }

        // Corners xz
        if (threadIdx.x < 1 && threadIdx.z < 1){
            f[i-1][j][k-1] = access_3d(fg, per_idx(gx-1, dim.x), gy, per_idx(gz-1, dim.z), dim.x, dim.y);
            f[i+blockDim.x][j][k+blockDim.z] = access_3d(fg, per_idx(gx+blockDim.x, dim.x), gy, per_idx(gz+blockDim.z, dim.z), dim.x, dim.y);
            f[i-1][j][k+blockDim.z] = access_3d(fg, per_idx(gx-1, dim.x), gy, per_idx(gz+blockDim.z, dim.z), dim.x, dim.y);
            f[i+blockDim.x][j][k-1] = access_3d(fg, per_idx(gx+blockDim.x, dim.x), gy, per_idx(gz-1, dim.z), dim.x, dim.y);
        }

        // Corners yz
        if (threadIdx.y < 1 && threadIdx.z < 1){
            f[i][j-1][k-1] = access_3d(fg, gx, per_idx(gy-1, dim.y), per_idx(gz-1, dim.z), dim.x, dim.y);
            f[i][j+blockDim.y][k+blockDim.z] = access_3d(fg, gx, per_idx(gy+blockDim.y, dim.y), per_idx(gz+blockDim.z, dim.z), dim.x, dim.y);
            f[i][j-1][k+blockDim.z] = access_3d(fg, gx, per_idx(gy-1, dim.y), per_idx(gz+blockDim.z, dim.z), dim.x, dim.y);
            f[i][j+blockDim.y][k-1] = access_3d(fg, gx, per_idx(gy+blockDim.y, dim.y), per_idx(gz-1, dim.z), dim.x, dim.y);
        }

        // Corners all
        if (threadIdx.x < 1 && threadIdx.y < 1 && threadIdx.z < 1){
            f[i-1][j-1][k-1] = access_3d(fg, per_idx(gx-1, dim.x), per_idx(gy-1, dim.y), per_idx(gz-1, dim.z), dim.x, dim.y);
            f[i+blockDim.x][j+blockDim.y][k+blockDim.z] = access_3d(fg, per_idx(gx+blockDim.x, dim.x), per_idx(gy+blockDim.y, dim.y), per_idx(gz+blockDim.z, dim.z), dim.x, dim.y);
            f[i-1][j+blockDim.y][k+blockDim.z] = access_3d(fg, per_idx(gx-1, dim.x), per_idx(gy+blockDim.y, dim.y), per_idx(gz+blockDim.z, dim.z), dim.x, dim.y);
            f[i+blockDim.x][j-1][k+blockDim.z] = access_3d(fg, per_idx(gx+blockDim.x, dim.x), per_idx(gy-1, dim.y), per_idx(gz+blockDim.z, dim.z), dim.x, dim.y);
            f[i+blockDim.x][j+blockDim.y][k-1] = access_3d(fg, per_idx(gx+blockDim.x, dim.x), per_idx(gy+blockDim.y, dim.y), per_idx(gz-1, dim.z), dim.x, dim.y);
            f[i+blockDim.x][j-1][k-1] = access_3d(fg, per_idx(gx+blockDim.x, dim.x), per_idx(gy-1, dim.y), per_idx(gz-1, dim.z), dim.x, dim.y);
            f[i-1][j+blockDim.y][k-1] = access_3d(fg, per_idx(gx-1, dim.x), per_idx(gy+blockDim.y, dim.y), per_idx(gz-1, dim.z), dim.x, dim.y);
            f[i-1][j-1][k+blockDim.z] = access_3d(fg, per_idx(gx-1, dim.x), per_idx(gy-1, dim.y), per_idx(gz+blockDim.z, dim.z), dim.x, dim.y);
        }

        // Sync
        __syncthreads();

        /* calculate some time savers */
        float3 two_h, h_sqr;

        two_h.x = 2.0f * h.x;
        two_h.y = 2.0f * h.y;
        two_h.z = 2.0f * h.z;
        h_sqr.x = h.x * h.x;
        h_sqr.y = h.y * h.y;
        h_sqr.z = h.z * h.z;
        float two_hx_hy = 2.0f * h.x * h.y;
        float two_hx_hz = 2.0f * h.x * h.z;
        float two_hy_hz = 2.0f * h.y * h.z;

        /* central spatial derivatives */
        float fx  = (f[i+1][j][k] - f[i-1][j][k]) / two_h.x;
        float fy  = (f[i][j+1][k] - f[i][j-1][k]) / two_h.y;
        float fz  = (f[i][j][k+1] - f[i][j][k-1]) / two_h.z;
        float fxx = (f[i+1][j][k] - 2.0f * f[i][j][k] + f[i-1][j][k]) / h_sqr.x;
        float fyy = (f[i][j+1][k] - 2.0f * f[i][j][k] + f[i][j-1][k]) / h_sqr.y;
        float fzz = (f[i][j][k+1] - 2.0f * f[i][j][k] + f[i][j][k-1]) / h_sqr.z;

        float fxy, fyz, fxz;

        if (fx * fy < 0.0f)
            fxy = (   f[i+1][j+1][k] - f[i][j+1][k] - f[i+1][j][k] + f[i][j][k]
                      + f[i-1][j-1][k] - f[i][j-1][k] - f[i-1][j][k] + f[i][j][k] )
                  / two_hx_hy;
        else
            fxy = ( - f[i-1][j+1][k] + f[i][j+1][k] + f[i+1][j][k] - f[i][j][k]
                    - f[i+1][j-1][k] + f[i][j-1][k] + f[i-1][j][k] - f[i][j][k] )
                  / two_hx_hy;

        if (fy * fz < 0.0f)
            fyz = (   f[i][j+1][k+1] - f[i][j+1][k] - f[i][j][k+1] + f[i][j][k]
                      + f[i][j-1][k-1] - f[i][j-1][k] - f[i][j][k-1] + f[i][j][k] )
                  / two_hy_hz;
        else
            fyz = ( - f[i][j+1][k-1] + f[i][j+1][k] + f[i][j][k+1] - f[i][j][k]
                    - f[i][j-1][k+1] + f[i][j-1][k] + f[i][j][k-1] - f[i][j][k] )
                  / two_hy_hz;

        if (fx * fz < 0.0f)
            fxz = (   f[i+1][j][k+1] - f[i+1][j][k] - f[i][j][k+1] + f[i][j][k]
                      + f[i-1][j][k-1] - f[i-1][j][k] - f[i][j][k-1] + f[i][j][k] )
                  / two_hx_hz;
        else
            fxz = ( - f[i-1][j][k+1] + f[i][j][k+1] + f[i+1][j][k] - f[i][j][k]
                    - f[i+1][j][k-1] + f[i][j][k-1] + f[i-1][j][k] - f[i][j][k] )
                  / two_hx_hz;



        float grad_sqr = (fx * fx + fy * fy + fz * fz);
        if (grad_sqr != 0.0) {
            access_3d(ug, gx, gy, gz, dim.x, dim.y) = f[i][j][k]
                                                      + ht * (fx * fx * (fyy + fzz) + fy * fy * (fxx + fzz) +
                                                              fz * fz * (fxx + fyy)
                                                              - 2.0f * fx * fy * fxy - 2.0f * fy * fz * fyz -
                                                              2.0f * fx * fz * fxz) / grad_sqr;
        }

    }

    __host__
    void mcm_CUDA
            (float    ht,        /* time step size, 0 < ht <= 0.25 */
             long     nx,        /* image dimension in x direction */
             long     ny,        /* image dimension in y direction */
             long     nz,        /* image dimension in z direction */
             float    hx,        /* pixel width in x direction */
             float    hy,        /* pixel width in y direction */
             float    hz,        /* pixel width in y direction */
             float    *d_u)      /* input: original image ;  output: smoothed */

    {
        float *d_f;
        size_t size = nx * ny * nz;
        size_t size_bytes = size * sizeof(float);

    /* ---- allocate storage f ---- */
        hipMalloc((void **) &d_f, size_bytes);

    /* ---- copy u into f ---- */
        hipMemcpy(d_f, d_u, size_bytes, hipMemcpyDeviceToDevice);

    /* loop */
        dim3 block;
        block.x = 8;
        block.y = 8;
        block.z = 8;

        dim3 grid;
        grid.x = nx / 8 + 1;
        grid.y = nx / 8 + 1;
        grid.z = nx / 8 + 1;

        dim3 dim;
        dim.x = nx;
        dim.y = ny;
        dim.z = nz;

        float3 h;
        h.x = hx;
        h.y = hy;
        h.z = hz;

        mcm_kernel<<<grid, block, 1000 * sizeof(float)>>>(d_f, d_u, ht, h, dim);

    /* ---- disallocate storage for f ---- */
        hipFree(d_f);
    }

    __host__
    void mcm_iterate_CUDA(float *d_u,
                          long pmax,
                          float ht,
                          int nx, int ny, int nz,
                          float hx, float hy, float hz,
                          Npp8u *d_minmax,
                          Npp8u *d_meastd,
                          int verbose)
    {
        float h_min, h_max, h_mean, h_std;

        if (verbose) {
            analyse_CUDA(d_u, nx, ny, nz, d_minmax, d_meastd, &h_min, &h_max, &h_mean, &h_std);
            printf("Input Data: min: %3.6f, max: %3.6f, mean: %3.6f, variance: %3.6f\n\n", h_min, h_max, h_mean, h_std * h_std);
        }

        for (long p=1; p<=pmax; p++) {
            /* perform one iteration */
            if(verbose) {
                printf("iteration number: %5ld / %ld \n", p, pmax);
            }
            mcm_CUDA(ht, nx, ny, nz, hx, hy, hz, d_u);

            /* check minimum, maximum, mean, variance */
            if (verbose) {
                analyse_CUDA(d_u, nx, ny, nz, d_minmax, d_meastd, &h_min, &h_max, &h_mean, &h_std);
                printf("min: %1.6f, max: %1.6f, mean: %1.6f, variance: %1.6f\n", h_min, h_max, h_mean, h_std*h_std);
            }
        }
    }

}
