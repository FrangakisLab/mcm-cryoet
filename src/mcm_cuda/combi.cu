#include "hip/hip_runtime.h"
#include "combi.cuh"

extern "C"{
    __global__
    void segm_combi_kernel(const float *fg, float *ug, float alpha, float beta, int3 dim) {
        // Shared memory for this block
        __shared__ float f[10][10][10];

        // Global idx
        const int gx = (int)(blockIdx.x * blockDim.x + threadIdx.x);
        const int gy = (int)(blockIdx.y * blockDim.y + threadIdx.y);
        const int gz = (int)(blockIdx.z * blockDim.z + threadIdx.z);

        // Threads larger than the volume+1 go.
        if (gx >= dim.x) return;
        if (gy >= dim.y) return;
        if (gz >= dim.z) return;

        // Local idx
        const unsigned int i = threadIdx.x + 1;
        const unsigned int j = threadIdx.y + 1;
        const unsigned int k = threadIdx.z + 1;

        // Incomplete block borders
        int3 blockMax = make_int3(8, 8, 8);
        if (gx == dim.x-1) blockMax.x = (int)threadIdx.x + 1;
        if (gy == dim.y-1) blockMax.y = (int)threadIdx.y + 1;
        if (gz == dim.z-1) blockMax.z = (int)threadIdx.z + 1;

        // Copy the data (direct match).
        f[i][j][k] = access_3d(fg, gx, gy, gz, dim.x, dim.y);

        // Helper
        int gxm1 = per_idx(gx - 1, dim.x);
        int gym1 = per_idx(gy - 1, dim.y);
        int gzm1 = per_idx(gz - 1, dim.z);

        int gxpb = per_idx(gx + blockMax.x, dim.x);
        int gypb = per_idx(gy + blockMax.y, dim.y);
        int gzpb = per_idx(gz + blockMax.z, dim.z);

        // Overlap x
        if (threadIdx.x < 1) {
            f[i - 1][j][k] = access_3d(fg, gxm1, gy, gz, dim.x, dim.y);
            f[i + blockMax.x][j][k] = access_3d(fg, gxpb, gy, gz, dim.x, dim.y);
        }

        // Overlap y
        if (threadIdx.y < 1) {
            f[i][j - 1][k] = access_3d(fg, gx, gym1, gz, dim.x, dim.y);
            f[i][j + blockMax.y][k] = access_3d(fg, gx, gypb, gz, dim.x, dim.y);
        }

        // Overlap z
        if (threadIdx.z < 1) {
            f[i][j][k - 1] = access_3d(fg, gx, gy, gzm1, dim.x, dim.y);
            f[i][j][k + blockMax.z] = access_3d(fg, gx, gy, gzpb, dim.x, dim.y);
        }

        // Corners xy
        if (threadIdx.x < 1 && threadIdx.y < 1) {
            f[i - 1][j - 1][k] = access_3d(fg, gxm1, gym1, gz, dim.x, dim.y);
            f[i + blockMax.x][j + blockMax.y][k] = access_3d(fg, gxpb, gypb, gz, dim.x, dim.y);
            f[i - 1][j + blockMax.y][k] = access_3d(fg, gxm1, gypb, gz, dim.x, dim.y);
            f[i + blockMax.x][j - 1][k] = access_3d(fg, gxpb, gym1, gz, dim.x, dim.y);
        }

        // Corners xz
        if (threadIdx.x < 1 && threadIdx.z < 1) {
            f[i - 1][j][k - 1] = access_3d(fg, gxm1, gy, gzm1, dim.x, dim.y);
            f[i + blockMax.x][j][k + blockMax.z] = access_3d(fg, gxpb, gy, gzpb, dim.x, dim.y);
            f[i - 1][j][k + blockMax.z] = access_3d(fg, gxm1, gy, gzpb, dim.x, dim.y);
            f[i + blockMax.x][j][k - 1] = access_3d(fg, gxpb, gy, gzm1, dim.x, dim.y);
        }

        // Corners yz
        if (threadIdx.y < 1 && threadIdx.z < 1) {
            f[i][j - 1][k - 1] = access_3d(fg, gx, gym1, gzm1, dim.x, dim.y);
            f[i][j + blockMax.y][k + blockMax.z] = access_3d(fg, gx, gypb, gzpb, dim.x, dim.y);
            f[i][j - 1][k + blockMax.z] = access_3d(fg, gx, gym1, gzpb, dim.x, dim.y);
            f[i][j + blockMax.y][k - 1] = access_3d(fg, gx, gypb, gzm1, dim.x, dim.y);
        }

        // Corners all
        if (threadIdx.x < 1 && threadIdx.y < 1 && threadIdx.z < 1) {
            f[i - 1][j - 1][k - 1] = access_3d(fg, gxm1, gym1, gzm1, dim.x, dim.y);
            f[i + blockMax.x][j + blockMax.y][k + blockMax.z] = access_3d(fg, gxpb, gypb, gzpb, dim.x, dim.y);
            f[i - 1][j + blockMax.y][k + blockMax.z] = access_3d(fg, gxm1, gypb, gzpb, dim.x, dim.y);
            f[i + blockMax.x][j - 1][k + blockMax.z] = access_3d(fg, gxpb, gym1, gzpb, dim.x, dim.y);
            f[i + blockMax.x][j + blockMax.y][k - 1] = access_3d(fg, gxpb, gypb, gzm1, dim.x, dim.y);
            f[i + blockMax.x][j - 1][k - 1] = access_3d(fg, gxpb, gym1, gzm1, dim.x, dim.y);
            f[i - 1][j + blockMax.y][k - 1] = access_3d(fg, gxm1, gypb, gzm1, dim.x, dim.y);
            f[i - 1][j - 1][k + blockMax.z] = access_3d(fg, gxm1, gym1, gzpb, dim.x, dim.y);
        }

        // Sync
        __syncthreads();

        // Compute curvature
        float f0_x = (f[i + 1][j][k] - f[i - 1][j][k]) / 2.f;
        float f0_y = (f[i][j + 1][k] - f[i][j - 1][k]) / 2.f;
        float f0_z = (f[i][j][k + 1] - f[i][j][k - 1]) / 2.f;
        float f0_xx = (f[i + 1][j][k] - 2.0f * f[i][j][k] + f[i - 1][j][k]);
        float f0_yy = (f[i][j + 1][k] - 2.0f * f[i][j][k] + f[i][j - 1][k]);
        float f0_zz = (f[i][j][k + 1] - 2.0f * f[i][j][k] + f[i][j][k - 1]);

        float f0_xy, f0_yz, f0_xz;

        if (f0_x * f0_y < 0.0)
            f0_xy = (f[i + 1][j + 1][k] - f[i][j + 1][k] - f[i + 1][j][k] + f[i][j][k]
                     + f[i - 1][j - 1][k] - f[i][j - 1][k] - f[i - 1][j][k] + f[i][j][k]) / 2.f;
        else
            f0_xy = (-f[i - 1][j + 1][k] + f[i][j + 1][k] + f[i + 1][j][k] - f[i][j][k]
                     - f[i + 1][j - 1][k] + f[i][j - 1][k] + f[i - 1][j][k] - f[i][j][k]) / 2.f;

        if (f0_y * f0_z < 0.0)
            f0_yz = (f[i][j + 1][k + 1] - f[i][j + 1][k] - f[i][j][k + 1] + f[i][j][k]
                     + f[i][j - 1][k - 1] - f[i][j - 1][k] - f[i][j][k - 1] + f[i][j][k]) / 2.f;
        else
            f0_yz = (-f[i][j + 1][k - 1] + f[i][j + 1][k] + f[i][j][k + 1] - f[i][j][k]
                     - f[i][j - 1][k + 1] + f[i][j - 1][k] + f[i][j][k - 1] - f[i][j][k]) / 2.f;

        if (f0_x * f0_z < 0.0)
            f0_xz = (f[i + 1][j][k + 1] - f[i + 1][j][k] - f[i][j][k + 1] + f[i][j][k]
                     + f[i - 1][j][k - 1] - f[i - 1][j][k] - f[i][j][k - 1] + f[i][j][k]) / 2.f;
        else
            f0_xz = (-f[i - 1][j][k + 1] + f[i][j][k + 1] + f[i + 1][j][k] - f[i][j][k]
                     - f[i + 1][j][k - 1] + f[i][j][k - 1] + f[i - 1][j][k] - f[i][j][k]) / 2.f;

        float grad_sqr = f0_x * f0_x + f0_y * f0_y + f0_z * f0_z;

        float curv = 0.0f;

        if (grad_sqr != 0.0f) {
            curv = 0.2f * (f0_x * f0_x * (f0_yy + f0_zz) + f0_y * f0_y * (f0_xx + f0_zz)
                           + f0_z * f0_z * (f0_xx + f0_yy) - 2.0f * f0_x * f0_y * f0_xy
                           - 2.0f * f0_y * f0_z * f0_yz - 2.0f * f0_x * f0_z * f0_xz) / grad_sqr;
        }

        float fm_x = f[i][j][k] - f[i - 1][j][k];
        float fp_x = f[i + 1][j][k] - f[i][j][k];
        float fm_y = f[i][j][k] - f[i][j - 1][k];
        float fp_y = f[i][j + 1][k] - f[i][j][k];
        float fm_z = f[i][j][k] - f[i][j][k - 1];
        float fp_z = f[i][j][k + 1] - f[i][j][k];

        float max_fm_x, min_fp_x, max_fm_y, min_fp_y, max_fm_z, min_fp_z;

        if (alpha > 0.f) {
            max_fm_x = min(fm_x, 0.f);
            min_fp_x = max(fp_x, 0.f);
            max_fm_y = min(fm_y, 0.f);
            min_fp_y = max(fp_y, 0.f);
            max_fm_z = min(fm_z, 0.f);
            min_fp_z = max(fp_z, 0.f);
        } else {
            max_fm_x = max(fm_x, 0.f);
            min_fp_x = min(fp_x, 0.f);
            max_fm_y = max(fm_y, 0.f);
            min_fp_y = min(fp_y, 0.f);
            max_fm_z = max(fm_z, 0.f);
            min_fp_z = min(fp_z, 0.f);
        }

        /* Level set in all directions */
        float lvset = 0.5f * sqrtf(max_fm_x * max_fm_x + min_fp_x * min_fp_x
                                   + max_fm_y * max_fm_y + min_fp_y * min_fp_y
                                   + max_fm_z * max_fm_z + min_fp_z * min_fp_z);

        // Return
        access_3d(ug, gx, gy, gz, dim.x, dim.y) = f[i][j][k] + beta * curv + alpha * lvset;
    }

    __host__
    void segm_combi_CUDA

            (int nx,        /* image dimension in x direction */
             int ny,        /* image dimension in y direction */
             int nz,        /* image dimension in z direction */
             float alpha,      /* Balance*/
             float beta,
             float *d_u)       /* input: original image ;  output: smoothed */

    {
        float *d_f;
        size_t size = nx * ny * nz;
        size_t size_bytes = size * sizeof(float);

    /* ---- allocate storage f ---- */
        CUDA_CALL(hipMalloc((void **) &d_f, size_bytes));

    /* ---- copy u into f ---- */
        CUDA_CALL(hipMemcpy(d_f, d_u, size_bytes, hipMemcpyDeviceToDevice));

    /* loop */
        dim3 block;
        block.x = 8;
        block.y = 8;
        block.z = 8;

        dim3 grid;
        grid.x = nx / 8 + 1;
        grid.y = ny / 8 + 1;
        grid.z = nz / 8 + 1;

        int3 dim;
        dim.x = nx;
        dim.y = ny;
        dim.z = nz;

        segm_combi_kernel<<<grid, block>>>(d_f, d_u, alpha, beta, dim);

    /* ---- disallocate storage for f ---- */
        CUDA_CALL(hipFree(d_f));
    }

    __host__
    void segm_combi_iterate_CUDA(float *d_u,
                                 int pmax,
                                 int nx, int ny, int nz,
                                 float alpha,
                                 float beta,
                                 Npp8u *d_minmax,
                                 Npp8u *d_meastd,
                                 int verbose) {
        float h_min, h_max, h_mean, h_std;

        if (verbose) {
            analyse_CUDA(d_u, nx, ny, nz, d_minmax, d_meastd, &h_min, &h_max, &h_mean, &h_std);
            printf("Input Data: min: %3.6f, max: %3.6f, mean: %3.6f, variance: %3.6f\n\n", h_min, h_max, h_mean,
                   h_std * h_std);
        }

        for (int p = 1; p <= pmax; p++) {
            /* perform one iteration */
            if (verbose) {
                printf("iteration number: %5d / %d \n", p, pmax);
            }
            segm_combi_CUDA(nx, ny, nz, alpha, beta, d_u);

            /* check minimum, maximum, mean, variance */
            if (verbose) {
                analyse_CUDA(d_u, nx, ny, nz, d_minmax, d_meastd, &h_min, &h_max, &h_mean, &h_std);
                printf("min: %1.6f, max: %1.6f, mean: %1.6f, variance: %1.6f\n", h_min, h_max, h_mean, h_std * h_std);
            }
        }
    }
}
